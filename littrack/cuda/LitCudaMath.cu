#include "hip/hip_runtime.h"
#ifndef LITCUDAMATH_H_
#define LITCUDAMATH_H_

#include "LitCudaTrackParam.h"
#include "LitCudaHit.h"
#include "LitCudaTrack.h"

__device__ float ChiSq(
   const LitCudaTrackParam* par,
   const LitCudaStripHit* hit)
{
   const float TWO = 2.;

   float duu = hit->Du * hit->Du;
   float phiCosSq = hit->phiCos * hit->phiCos;
   float phiSinSq = hit->phiSin * hit->phiSin;
   float phi2SinCos = TWO * hit->phiCos * hit->phiSin;

   float r = hit->U - par->X * hit->phiCos - par->Y * hit->phiSin;
   float rr = r * r;
   float norm = duu + par->C0 * phiCosSq + phi2SinCos * par->C1 + par->C5 * phiSinSq;
// float norm = duu + par->C0 * hit->phiCos + par->C5 * hitphiSin;

   return rr / norm;
}

__device__ float ChiSq(
   const LitCudaTrackParam* par,
   const LitCudaPixelHit* hit)
{
   const float TWO = 2.;

   float dxx = hit->Dx * hit->Dx;
   float dyy = hit->Dy * hit->Dy;
   float dx = hit->X - par->X;
   float dy = hit->Y - par->Y;

   float norm = -dxx * dyy + dxx * par->C5 + dyy * par->C0 - par->C0 * par->C5 + hit->Dxy * hit->Dxy - TWO * hit->Dxy * par->C1 + par->C1 * par->C1;
// if (norm == 0->) norm = 1e-10;

   return (-dx * dx * (dyy - par->C5) - dy * dy * (dxx - par->C0) + TWO * dx * dy * (hit->Dxy - par->C1)) / norm;
}

__device__ unsigned short NDF(
   const LitCudaTrack* track)
{
   // TODO check NDF

   unsigned short ndf =  2 * track->nofHits - 5;
   if (ndf > 0) { return ndf; }
   else { return 1; }

// int ndf = 0;
// for (int i = 0; i < track->GetNofHits(); i++){
//    if (track->GetHit(i)->GetType() == kLITPIXELHIT) ndf += 2;
//    else if (track->GetHit(i)->GetType() == kLITSTRIPHIT) ndf++;
// }
// ndf -= 5;
// if (ndf > 0) return ndf; else return 1;
}
//#undef cnst
#endif /* LITCUDAMATH_H_ */
