#include "hip/hip_runtime.h"
#include "LitCudaTrackFitter.cu"
#include "LitCudaDetectorGeometry.h"

__global__ void LitCudaFitTrack(
   LitCudaTrack* tracks[],
   const LitCudaDetectorLayout* layout)
{
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   LitCudaTrack* track = tracks[i];
   LitCudaTrackFitter(track, layout);
}

//Host code
extern "C" void LitCudaFitTracks(
   LitCudaTrack* h_tracks[],
   const LitCudaDetectorLayout* h_layout,
   int nofTracks)
{
//    printf("Executing FitTracks...\n");

   // Calculate required memory size
   size_t size = nofTracks * sizeof(LitCudaTrack);
   size_t layoutSize = sizeof(LitCudaDetectorLayout);

//    printf("Number of elements %d. Size %d \n", nofTracks, size);

   // Allocate device memory
//    printf("Allocate device memory...\n");
   LitCudaTrack** d_tracks;
   hipMalloc((void**)&d_tracks, size);

   LitCudaDetectorLayout* d_layout;
   hipMalloc((void**)&d_layout, layoutSize);

   //Copy vectors from host to device memory
//    printf("Copy vectors from host to device memory...\n");
   hipMemcpy(d_tracks, h_tracks, size, hipMemcpyHostToDevice);
   // Copy detector layout to device
   hipMemcpy(d_layout, h_layout, layoutSize, hipMemcpyHostToDevice);

   //Invoke kernel
   int threadsPerBlock = 256;
   int blocksPerGrid = (nofTracks + threadsPerBlock - 1) / threadsPerBlock;
   LitCudaFitTrack<<<blocksPerGrid, threadsPerBlock>>>(d_tracks, d_layout);

   hipMemcpy(h_tracks, d_tracks, size, hipMemcpyDeviceToHost);

   //Free device memory
   hipFree(d_tracks);

//    printf("Finish...");
}
