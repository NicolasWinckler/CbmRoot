#include "hip/hip_runtime.h"
#ifndef LITCUDATRACKING_H_
#define LITCUDATRACKING_H_

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

#include "LitCudaHit.h"
#include "LitCudaTrack.h"
#include "LitCudaMaterialInfo.h"
#include "LitCudaField.cu"


//#ifndef LITCUDAADDMATERIAL_H_
//#define LITCUDAADDMATERIAL_H_

//#include "LitCudaTrackParam.h"
//#include "LitCudaMaterialInfo.h"

/* Function calculates effects from the material on the
 * track parameters and its covariance matrix.
 * @param par Pointer to INPUT/OUTPUT track parameters.
 * @param mat Pointer to material
 */
__device__ void LitCudaAddMaterial(
   LitCudaTrackParam* par,
   const LitCudaMaterialInfo* mat)
{
   const float ZERO = 0.0, ONE = 1., TWO = 2.;
   const float mass = 0.105658369; // muon mass [GeV/c]
   const float massSq = 0.105658369 * 0.105658369; // muon mass squared
   const float C1_2 = 0.5, C1_3 = 1./3.;
   const float me = 0.000511; // Electron mass [GeV/c]
   const float ratio = me / mass;

   float p = copysignf(1./par->Qp, par->Qp); // Momentum [GeV/c]
   float E = sqrtf(massSq + p * p);
   float beta = p / E;
   float betaSq = beta * beta;
   float gamma = E / mass;
   float gammaSq = gamma * gamma;

   //scale material thickness
   float norm = sqrtf(ONE + par->Tx * par->Tx + par->Ty * par->Ty);
   float thickness = norm * mat->Thickness;
   float radThick = thickness / mat->X0;
   float sqrtRadThick = sqrtf(radThick);//mat.SqrtRadThick;
   float logRadThick = logf(radThick);//mat.LogRadThick;

   /*
    * Energy loss corrections
    */

   // Bethe-Block
   const float K = 0.000307075; // GeV * g^-1 * cm^2
   float Tmax = (2 * me * betaSq * gammaSq) / (ONE + TWO * gamma * ratio + ratio * ratio);

   // density correction
   float dc = ZERO;
// if (p > 0.5) { // for particles above 1 Gev
   const float c7 = 28.816;
   const float c8 = 1e-9;
   float hwp = c7 * sqrtf(mat->Rho * mat->Z / mat->A) * c8; // GeV
   dc = logf(hwp/mat->I) + logf(beta*gamma) - C1_2;
// }

   float bbLoss = K * (mat->Z / mat->A) * (1./betaSq) *
                  (C1_2 * logf(TWO * me * betaSq * gammaSq * Tmax / (mat->I * mat->I)) - betaSq - dc);

   // Bethe-Heitler
// float bhLoss = (E * ratio * ratio)/(mat.X0 * mat.Rho);
   float bhLoss = ZERO;

   // Pair production approximation
// static const float c3 = 7e-5;
// float ppLoss = c3 * E / (mat.X0 * mat.Rho);
   float ppLoss = ZERO;

   // Integrated value of the energy loss
   float energyLoss = (bbLoss + bhLoss + ppLoss) * mat->Rho * thickness;

   // Correct Q/p value due to energy loss
   float Enew = E - energyLoss;
   float pnew = sqrtf(Enew * Enew - massSq);
   par->Qp = copysignf(1./pnew, par->Qp);

   // Calculate Q/p correction in the covariance matrix
   float betanew = pnew / Enew;
   float betaSqnew = betanew * betanew;
   float gammanew = Enew / mass;
// float gammaSqnew = gammanew * gammanew;

   // Calculate xi factor (KeV).
   const float c4 = 153.5;
   float XI = (c4 * mat->Z * thickness * mat->Rho)/(mat->A * betaSqnew);

   // Maximum energy transfer to atomic electron (KeV).
   float etanew = betanew * gammanew;
   float etaSqnew = etanew * etanew;
   float F1 = TWO * me * etaSqnew;
   float F2 = ONE + TWO * ratio * gammanew + ratio * ratio;
   const float c5 = 1e6;
   float emax = c5 * F1/F2;

   const float c6 = 1e-12;
   float dedxSq = XI * emax * (ONE - C1_2 * betaSqnew) * c6;

   float p2 = pnew * pnew;
   float p6 = p2 * p2 * p2;
   float qpCorr = (Enew * Enew * dedxSq) / p6;
   par->C14 += qpCorr;
   // end calculate Q/p correction in the covariance matrix

   /*
    * End energy loss corrections
    */

   /*
    * Multiple scattering corrections
    */
   float tx = par->Tx;
   float ty = par->Ty;
   float bcp = betanew * pnew;
   const float c1 = 0.0136, c2 = 0.038;
   float theta = c1 * (1./bcp) * sqrtRadThick * (ONE + c2 * logRadThick);
   float thetaSq = theta * theta;

   float t = ONE + tx * tx + ty * ty;

   float Q33 = (1 + tx * tx) * t * thetaSq;
   float Q44 = (1 + ty * ty) * t * thetaSq;
   float Q34 = tx * ty * t * thetaSq;

   float T23 = thickness * thickness * C1_3;
   float T2 = thickness * C1_2;

   par->C0 += Q33 * T23;
   par->C1 += Q34 * T23;
   par->C2 += Q33 * T2;
   par->C3 += Q34 * T2;

   par->C5 += Q44 * T23;
   par->C6 += Q34 * T2;
   par->C7 += Q44 * T2;

   par->C9 += Q33;
   par->C10 += Q34;

   par->C12 += Q44;

   /*
    * End multiple scattering corrections
    */
}

//#endif /* LITCUDAADDMATERIAL_H_ */




//#ifndef LITCUDAEXTRAPOLATION_H_
//#define LITCUDAEXTRAPOLATION_H_

//#include "LitCudaTrackParam.h"
//#include "LitCudaField.h"

/* Line track extrapolation for CUDA version of the tracking.
 *@param par Input/Output track parameters.
 *@param zOut Z position to extrapolate to.
 */
__device__ void LitCudaLineExtrapolation(
   LitCudaTrackParam* par,
   float zOut)
{
   float dz = zOut - par->Z;

   // transport state vector F*X*F.T()
   par->X += dz * par->Tx;
   par->Y += dz * par->Ty;

   // transport covariance matrix F*C*F.T()
   float t3 = par->C2 + dz * par->C9;
   float t7 = dz * par->C10;
   float t8 = par->C3 + t7;
   float t19 = par->C7 + dz * par->C12;
   par->C0 += dz * par->C2 + t3 * dz;
   par->C1 += dz * par->C6 + t8 * dz;
   par->C2 = t3;
   par->C3 = t8;
   par->C4 += dz * par->C11;
   par->C5 += dz * par->C7 + t19 * dz;
   par->C6 += t7;
   par->C7 = t19;
   par->C8 += dz * par->C13;

   par->Z = zOut;
}

/* Runge-Kutta track extrapolation for parallel version of the tracking.
 *@param par Input/Output track parameters.
 *@param zOut Z position to extrapolate to.
 */
__device__ void LitCudaRK4Extrapolation(
   LitCudaTrackParam* par,
   float zOut,
   const LitCudaFieldRegion* field)
{
   const float fC = 0.000299792458;
   const float ZERO = 0., ONE = 1., TWO = 2., C1_3 = 1./3., C1_6 = 1./6.;

   float coef[4] = {0.0, 0.5, 0.5, 1.0};

   float Ax[4], Ay[4];
   float dAx_dtx[4], dAy_dtx[4], dAx_dty[4], dAy_dty[4];
   float kx[4];
   float ky[4];
   float ktx[4];
   float kty[4];

   float zIn = par->Z;
   float h = zOut - zIn;
   float hC   = h * fC;
   float hCqp = h * fC * par->Qp;
   float x0[4];

   float x[4] = {par->X, par->Y, par->Tx, par->Ty};

   float F[25]; // derivatives, transport matrix

   // Get field values
   LitCudaFieldValue B[4];
   GetFieldValue(zIn + coef[0] * h, field, &B[0]);
   GetFieldValue(zIn + coef[1] * h, field, &B[1]);
   B[2] = B[1];
   GetFieldValue(zIn + coef[3] * h, field, &B[3]);

   // Calculation for zero step
   {
      float Bx = B[0].Bx;
      float By = B[0].By;
      float Bz = B[0].Bz;

      float tx = x[2];
      float ty = x[3];
      float txtx = tx * tx;
      float tyty = ty * ty;
      float txty = tx * ty;
      float txtxtyty1 = ONE + txtx + tyty;
      float t1 = sqrtf(txtxtyty1);
      float t2 = (1./txtxtyty1); //1.0 / txtxtyty1;

      Ax[0] = ( txty * Bx + ty * Bz - ( ONE + txtx ) * By ) * t1;
      Ay[0] = (-txty * By - tx * Bz + ( ONE + tyty ) * Bx ) * t1;

      dAx_dtx[0] = Ax[0] * tx * t2 + ( ty * Bx - TWO * tx * By ) * t1;
      dAx_dty[0] = Ax[0] * ty * t2 + ( tx * Bx + Bz ) * t1;
      dAy_dtx[0] = Ay[0] * tx * t2 + (-ty * By - Bz ) * t1;
      dAy_dty[0] = Ay[0] * ty * t2 + (-tx * By + TWO * ty * Bx ) * t1;

      kx[0] = tx * h;
      ky[0] = ty * h;
      ktx[0] = Ax[0] * hCqp;
      kty[0] = Ay[0] * hCqp;
   }
   // end calculation for zero step

   // Calculate for steps starting from 1
   for (unsigned int iStep = 1; iStep < 4; iStep++) { // 1
      x[0] = par->X  + coef[iStep] * kx[iStep - 1];
      x[1] = par->Y  + coef[iStep] * ky[iStep - 1];
      x[2] = par->Tx + coef[iStep] * ktx[iStep - 1];
      x[3] = par->Ty + coef[iStep] * kty[iStep - 1];

      float Bx = B[iStep].Bx;
      float By = B[iStep].By;
      float Bz = B[iStep].Bz;

      float tx = x[2];
      float ty = x[3];
      float txtx = tx * tx;
      float tyty = ty * ty;
      float txty = tx * ty;
      float txtxtyty1 = ONE + txtx + tyty;
      float t1 = sqrtf(txtxtyty1);
      float t2 = (1./txtxtyty1); //1.0 / txtxtyty1;

      Ax[iStep] = ( txty * Bx + ty * Bz - ( ONE + txtx ) * By ) * t1;
      Ay[iStep] = (-txty * By - tx * Bz + ( ONE + tyty ) * Bx ) * t1;

      dAx_dtx[iStep] = Ax[iStep] * tx * t2 + ( ty * Bx - TWO * tx * By ) * t1;
      dAx_dty[iStep] = Ax[iStep] * ty * t2 + ( tx * Bx + Bz ) * t1;
      dAy_dtx[iStep] = Ay[iStep] * tx * t2 + (-ty * By - Bz ) * t1;
      dAy_dty[iStep] = Ay[iStep] * ty * t2 + (-tx * By + TWO * ty * Bx ) * t1;

      kx[iStep] = tx * h;
      ky[iStep] = ty * h;
      ktx[iStep] = Ax[iStep] * hCqp;
      kty[iStep] = Ay[iStep] * hCqp;

   } // 1

   // Calculate output state vector
//   for (unsigned int i = 0; i < 4; i++) xOut[i] = CalcOut(xIn[i], k[i]);
   par->X  += kx[0] * C1_6 + kx[1] * C1_3 + kx[2] * C1_3 + kx[3] * C1_6;
   par->Y  += ky[0] * C1_6 + ky[1] * C1_3 + ky[2] * C1_3 + ky[3] * C1_6;
   par->Tx += ktx[0] * C1_6 + ktx[1] * C1_3 + ktx[2] * C1_3 + ktx[3] * C1_6;
   par->Ty += kty[0] * C1_6 + kty[1] * C1_3 + kty[2] * C1_3 + kty[3] * C1_6;
//   xOut[4] = xIn[4];


   // Calculation of the derivatives

   // derivatives dx/dx and dx/dy
   // dx/dx
   F[0] = ONE;
   F[5] = ZERO;
   F[10] = ZERO;
   F[15] = ZERO;
   F[20] = ZERO;
   // dx/dy
   F[1] = ZERO;
   F[6] = ONE;
   F[11] = ZERO;
   F[16] = ZERO;
   F[21] = ZERO;
   // end of derivatives dx/dx and dx/dy

   // Derivatives dx/tx
   x[0] = x0[0] = ZERO;
   x[1] = x0[1] = ZERO;
   x[2] = x0[2] = ONE;
   x[3] = x0[3] = ZERO;

   //Calculate for zero step
   kx[0] = x[2] * h;
   ky[0] = x[3] * h;
   //ktx[0] = (dAx_dtx[iStep] * x[2] + dAx_dty[iStep] * x[3]) * hCqp;
   kty[0] = (dAy_dtx[0] * x[2] + dAy_dty[0] * x[3]) * hCqp;
   // Calculate for steps starting from 1
   for (unsigned int iStep = 1; iStep < 4; iStep++) { // 2
      x[0] = x0[0] + coef[iStep] * kx[iStep - 1];
      x[1] = x0[1] + coef[iStep] * ky[iStep - 1];
//         x[2] = x0[2] + coef[iStep] * ktx[iStep - 1];
      x[3] = x0[3] + coef[iStep] * kty[iStep - 1];

      kx[iStep] = x[2] * h;
      ky[iStep] = x[3] * h;
      //ktx[iStep] = (dAx_dtx[iStep] * x[2] + dAx_dty[iStep] * x[3]) * hCqp;
      kty[iStep] = (dAy_dtx[iStep] * x[2] + dAy_dty[iStep] * x[3]) * hCqp;
   } // 2

   F[2]  = x0[0] + kx[0] * C1_6 + kx[1] * C1_3 + kx[2] * C1_3 + kx[3] * C1_6;
   F[7]  = x0[1] + ky[0] * C1_6 + ky[1] * C1_3 + ky[2] * C1_3 + ky[3] * C1_6;
   F[12] = ONE;
   F[17] = x0[3] + kty[0] * C1_6 + kty[1] * C1_3 + kty[2] * C1_3 + kty[3] * C1_6;
   F[22] = ZERO;
   // end of derivatives dx/dtx

   // Derivatives    dx/ty
   x[0] = x0[0] = ZERO;
   x[1] = x0[1] = ZERO;
   x[2] = x0[2] = ZERO;
   x[3] = x0[3] = ONE;

   //Calculate for zero step
   kx[0] = x[2] * h;
   ky[0] = x[3] * h;
   ktx[0] = (dAx_dtx[0] * x[2] + dAx_dty[0] * x[3]) * hCqp;
   //kty[0] = (dAy_dtx[iStep] * x[2] + dAy_dty[iStep] * x[3]) * hCqp;
   //Calculate for steps starting from 1
   for (unsigned int iStep = 1; iStep < 4; iStep++) { // 4
      x[0] = x0[0] + coef[iStep] * kx[iStep - 1];
      x[1] = x0[1] + coef[iStep] * ky[iStep - 1];
      x[2] = x0[2] + coef[iStep] * ktx[iStep - 1];
//         x[3] = x0[0] + coef[iStep] * kty[iStep - 1];

      kx[iStep] = x[2] * h;
      ky[iStep] = x[3] * h;
      ktx[iStep] = (dAx_dtx[iStep] * x[2] + dAx_dty[iStep] * x[3]) * hCqp;
      //kty[iStep] = (dAy_dtx[iStep] * x[2] + dAy_dty[iStep] * x[3]) * hCqp;
   }  // 4

   F[3]  = x0[0] + kx[0] * C1_6 + kx[1] * C1_3 + kx[2] * C1_3 + kx[3] * C1_6;
   F[8]  = x0[1] + ky[0] * C1_6 + ky[1] * C1_3 + ky[2] * C1_3 + ky[3] * C1_6;
   F[13] = x0[2] + ktx[0] * C1_6 + ktx[1] * C1_3 + ktx[2] * C1_3 + ktx[3] * C1_6;
   F[18] = ONE;
   F[23] = ZERO;
   // end of derivatives dx/dty

   // Derivatives dx/dqp
   x[0] = x0[0] = ZERO;
   x[1] = x0[1] = ZERO;
   x[2] = x0[2] = ZERO;
   x[3] = x0[3] = ZERO;

   //Calculate for zero step
   kx[0] = x[2] * h;
   ky[0] = x[3] * h;
   ktx[0] = Ax[0] * hC + hCqp * (dAx_dtx[0] * x[2] + dAx_dty[0] * x[3]);
   kty[0] = Ay[0] * hC + hCqp * (dAy_dtx[0] * x[2] + dAy_dty[0] * x[3]);
   //Calculate for steps starting from 1
   for (unsigned int iStep = 1; iStep < 4; iStep++) { // 4
      x[0] = x0[0] + coef[iStep] * kx[iStep - 1];
      x[1] = x0[1] + coef[iStep] * ky[iStep - 1];
      x[2] = x0[2] + coef[iStep] * ktx[iStep - 1];
      x[3] = x0[3] + coef[iStep] * kty[iStep - 1];

      kx[iStep] = x[2] * h;
      ky[iStep] = x[3] * h;
      ktx[iStep] = Ax[iStep] * hC + hCqp * (dAx_dtx[iStep] * x[2] + dAx_dty[iStep] * x[3]);
      kty[iStep] = Ay[iStep] * hC + hCqp * (dAy_dtx[iStep] * x[2] + dAy_dty[iStep] * x[3]);
   }  // 4

   F[4]  = x0[0] + kx[0] * C1_6 + kx[1] * C1_3 + kx[2] * C1_3 + kx[3] * C1_6;
   F[9]  = x0[1] + ky[0] * C1_6 + ky[1] * C1_3 + ky[2] * C1_3 + ky[3] * C1_6;
   F[14] = x0[2] + ktx[0] * C1_6 + ktx[1] * C1_3 + ktx[2] * C1_3 + ktx[3] * C1_6;
   F[19] = x0[3] + kty[0] * C1_6 + kty[1] * C1_3 + kty[2] * C1_3 + kty[3] * C1_6;
   F[24] = 1.;
   // end of derivatives dx/dqp

   // end calculation of the derivatives


   // Transport C matrix
   {
      float cIn[15] = {par->C0,  par->C1,  par->C2,  par->C3,  par->C4,
                       par->C5,  par->C6,  par->C7,  par->C8,  par->C9,
                       par->C10, par->C11, par->C12, par->C13, par->C14
                      };
      // F*C*Ft
      float A = cIn[2] + F[2] * cIn[9] + F[3] * cIn[10] + F[4] * cIn[11];
      float B = cIn[3] + F[2] * cIn[10] + F[3] * cIn[12] + F[4] * cIn[13];
      float C = cIn[4] + F[2] * cIn[11] + F[3] * cIn[13] + F[4] * cIn[14];

      float D = cIn[6] + F[7] * cIn[9] + F[8] * cIn[10] + F[9] * cIn[11];
      float E = cIn[7] + F[7] * cIn[10] + F[8] * cIn[12] + F[9] * cIn[13];
      float G = cIn[8] + F[7] * cIn[11] + F[8] * cIn[13] + F[9] * cIn[14];

      float H = cIn[9] + F[13] * cIn[10] + F[14] * cIn[11];
      float I = cIn[10] + F[13] * cIn[12] + F[14] * cIn[13];
      float J = cIn[11] + F[13] * cIn[13] + F[14] * cIn[14];

      float K = cIn[13] + F[17] * cIn[11] + F[19] * cIn[14];

      par->C0 = cIn[0] + F[2] * cIn[2] + F[3] * cIn[3] + F[4] * cIn[4] + A * F[2] + B * F[3] + C * F[4];
      par->C1 = cIn[1] + F[2] * cIn[6] + F[3] * cIn[7] + F[4] * cIn[8] + A * F[7] + B * F[8] + C * F[9];
      par->C2 = A + B * F[13] + C * F[14];
      par->C3 = B + A * F[17] + C * F[19];
      par->C4 = C;

      par->C5 = cIn[5] + F[7] * cIn[6] + F[8] * cIn[7] + F[9] * cIn[8] + D * F[7] + E * F[8] + G * F[9];
      par->C6 = D + E * F[13] + G * F[14];
      par->C7 = E + D * F[17] + G * F[19];
      par->C8 = G;

      par->C9 = H + I * F[13] + J * F[14];
      par->C10 = I + H * F[17] + J * F[19];
      par->C11 = J;

      par->C12 = cIn[12] + F[17] * cIn[10] + F[19] * cIn[13] + (F[17] * cIn[9] + cIn[10] + F[19] * cIn[11]) * F[17] + K * F[19];
      par->C13 = K;

      par->C14 = cIn[14];
   }
   //end transport C matrix

   par->Z = zOut;
}

//#undef cnst
//#endif



//#ifndef LITCUDAFILTRATION_H_
//#define LITCUDAFILTRATION_H_

//#include "LitCudaTypes.h"
//#include "LitCudaHit.h"

//#define cnst static const fvec

__device__ void LitCudaFiltration (
   LitCudaTrackParam* par,
   const LitCudaPixelHit* hit)
{
   const float ONE = 1., TWO = 2.;

   float dxx = hit->Dx * hit->Dx;
   float dxy = hit->Dxy;
   float dyy = hit->Dy * hit->Dy;

   // calculate residuals
   float dx = hit->X - par->X;
   float dy = hit->Y - par->Y;

   // Calculate and inverse residual covariance matrix
   float t = ONE / (dxx * dyy + dxx * par->C5 + dyy * par->C0 + par->C0 * par->C5 -
                    dxy * dxy - TWO * dxy * par->C1 - par->C1 * par->C1);
   float R00 = (dyy + par->C5) * t;
   float R01 = -(dxy + par->C1) * t;
   float R11 = (dxx + par->C0) * t;

   // Calculate Kalman gain matrix
   float K00 = par->C0 * R00 + par->C1 * R01;
   float K01 = par->C0 * R01 + par->C1 * R11;
   float K10 = par->C1 * R00 + par->C5 * R01;
   float K11 = par->C1 * R01 + par->C5 * R11;
   float K20 = par->C2 * R00 + par->C6 * R01;
   float K21 = par->C2 * R01 + par->C6 * R11;
   float K30 = par->C3 * R00 + par->C7 * R01;
   float K31 = par->C3 * R01 + par->C7 * R11;
   float K40 = par->C4 * R00 + par->C8 * R01;
   float K41 = par->C4 * R01 + par->C8 * R11;

   // Calculate filtered state vector
   par->X  += K00 * dx + K01 * dy;
   par->Y  += K10 * dx + K11 * dy;
   par->Tx += K20 * dx + K21 * dy;
   par->Ty += K30 * dx + K31 * dy;
   par->Qp += K40 * dx + K41 * dy;

   // Calculate filtered covariance matrix
   float cIn[15] = {par->C0,  par->C1,  par->C2,  par->C3,  par->C4,
                    par->C5,  par->C6,  par->C7,  par->C8,  par->C9,
                    par->C10, par->C11, par->C12, par->C13, par->C14
                   };

   par->C0  += -K00 * cIn[0] - K01 * cIn[1];
   par->C1  += -K00 * cIn[1] - K01 * cIn[5];
   par->C2  += -K00 * cIn[2] - K01 * cIn[6];
   par->C3  += -K00 * cIn[3] - K01 * cIn[7];
   par->C4  += -K00 * cIn[4] - K01 * cIn[8];

   par->C5  += -K11 * cIn[5] - K10 * cIn[1];
   par->C6  += -K11 * cIn[6] - K10 * cIn[2];
   par->C7  += -K11 * cIn[7] - K10 * cIn[3];
   par->C8  += -K11 * cIn[8] - K10 * cIn[4];

   par->C9  += -K20 * cIn[2] - K21 * cIn[6];
   par->C10 += -K20 * cIn[3] - K21 * cIn[7];
   par->C11 += -K20 * cIn[4] - K21 * cIn[8];

   par->C12 += -K30 * cIn[3] - K31 * cIn[7];
   par->C13 += -K30 * cIn[4] - K31 * cIn[8];

   par->C14 += -K40 * cIn[4] - K41 * cIn[8];
}

__device__ void LitFiltration(
   LitCudaTrackParam* par,
   const LitCudaStripHit* hit)
{
   const float ONE = 1., TWO = 2.;

   float duu = hit->Du * hit->Du;
   float phiCosSq = hit->phiCos * hit->phiCos;
   float phiSinSq = hit->phiSin * hit->phiSin;
   float phi2SinCos = TWO * hit->phiCos * hit->phiSin;

   // residual
   float r = hit->U - par->C0 * hit->phiCos - par->C1 * hit->phiSin;
   float norm = duu + par->C0 * phiCosSq + phi2SinCos * par->C1 + par->C5 * phiSinSq;
   //  myf norm = duu + cIn[0] * phiCos + cIn[5] * phiSin;
   float R = (ONE/norm);

   // Calculate Kalman gain matrix
   float K0 = par->C0 * hit->phiCos + par->C1 * hit->phiSin;
   float K1 = par->C1 * hit->phiCos + par->C5 * hit->phiSin;
   float K2 = par->C2 * hit->phiCos + par->C6 * hit->phiSin;
   float K3 = par->C3 * hit->phiCos + par->C7 * hit->phiSin;
   float K4 = par->C4 * hit->phiCos + par->C8 * hit->phiSin;

   float KR0 = K0 * R;
   float KR1 = K1 * R;
   float KR2 = K2 * R;
   float KR3 = K3 * R;
   float KR4 = K4 * R;

   // Calculate filtered state vector
   par->X += KR0 * r;
   par->Y += KR1 * r;
   par->Tx += KR2 * r;
   par->Ty += KR3 * r;
   par->Qp += KR4 * r;

   // Calculate filtered covariance matrix
   par->C0 -= KR0 * K0;
   par->C1 -= KR0 * K1;
   par->C2 -= KR0 * K2;
   par->C3 -= KR0 * K3;
   par->C4 -= KR0 * K4;

   par->C5 -= KR1 * K1;
   par->C6 -= KR1 * K2;
   par->C7 -= KR1 * K3;
   par->C8 -= KR1 * K4;

   par->C9 -= KR2 * K2;
   par->C10 -= KR2 * K3;
   par->C11 -= KR2 * K4;

   par->C12 -= KR3 * K3;
   par->C13 -= KR3 * K4;

   par->C14 -= KR4 * K4;
}

//#undef cnst
//#endif /* LITCUDAFILTRATION_H_ */



#endif /* LITCUDATRACKING_H_ */



