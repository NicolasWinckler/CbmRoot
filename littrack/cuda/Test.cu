#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

#include "LitCudaHit.h"
#include "LitCudaMaterialInfo.h"

//Device code
__global__ void VecAdd(float* A, float* B, float* C)
{
   int i = blockDim.x * blockIdx.x + threadIdx.x;
//    if (i < N)
   C[i] = A[i] + B[i];
}

//Host code
extern "C" void AddVec(void)
{
   LitCudaPixelHit pixelHit;
   pixelHit.X = 10.;
//    LitCudaStripHit<float> stripHit;
//    stripHit.U = 20.;
   std::cout << "PLUS" << pixelHit << std::endl;


   printf("Executing AddVec...\n");
   const int N = 10000000;
   size_t size = N * sizeof(float);
   printf("Number of elements %d. Size %d \n", N, size);

   // Allocate vectors in host memory
   printf("Allocate arrays in host memory...\n");
   float* h_A = (float*) malloc(size);
   float* h_B = (float*) malloc(size);
   float* h_C = (float*) malloc(size);

   // Allocate vectors in device memory
   printf("Allocate arrays in device memory...\n");
   float* d_A;
   hipMalloc((void**)&d_A, size);
   float* d_B;
   hipMalloc((void**)&d_B, size);
   float* d_C;
   hipMalloc((void**)&d_C, size);

   //Copy vectors from host to device memory
   printf("Copy vectors from host to device memory...\n");
   hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

   //Invoke kernel
   int threadsPerBlock = 256;
   int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
   VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);

   hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

   //Free device memory
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   printf("Finish...");
}
